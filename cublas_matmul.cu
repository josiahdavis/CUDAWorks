#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void printMatrix(const float* matrix, int rows, int cols, const char* name){
    printf("Matrix %s showing top-left 5x5", name);
    for (int i = 0; i < 5 && i < rows; i++){
        for (int j = 0; j < 5 && j < cols; j++){
            printf("%5.2d ", matrix[i * cols + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void matrixMultiplyCublas(const int M, const int N, const int K){
    // Allocate host memory
    // M = the number of rows in C and A
    // N = the number of cols in C and B
    // K = the inner dimension, the columns in A and rows in B
    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];

    // Initialize matrices
    for (int i = 0; i < M * K; i++){
        h_A[i] = i;
        // h_A[i] = (float)rand()/(float)RAND_MAX;
    }
    for (int i = 0; i < K * N; i++){
        h_B[i] = i * 2;
        // h_B[i] = (float)rand()/(float)RAND_MAX;
    }
    
    // Print input matrices
    printMatrix(h_A, M, K, "A");
    printMatrix(h_B, K, N, "B");

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, M*K*sizeof(float));
    hipMalloc((void**)&d_B, K*N*sizeof(float));
    hipMalloc((void**)&d_C, M*N*sizeof(float));

    // Copy data to device
    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);

    // cuBLAS handle is required to use cuBLAS. It initializes the library 
    // and acts as a context for making cuBLAS function calls.
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f, beta = 0.0f; // C = α*(A@B)+β*C, not sure what the purpose of beta param
    // Compute C = A x B
    // CUBLAS_OP_N: is the non-transpose operation is selected
    // M, N: The number of rows and cols in C
    // K: The inner dimension of A and B
    // leading dimension (lda, ldb, ldc) is not necessarily the number of rows—it's the stride (or pitch) between consecutive columns).
    // cuBLAS stores matrices in column-major order, meaning that elements of each column are stored contiguously in memory.
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_A, M, d_B, K, &beta, d_C, M); 

    // Copy results back to host
    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    
    // Print output matrix
    printMatrix(h_C, M, N, "C");

    // Clean up
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
}

int main() {
    int M = 2;
    int K = 3;
    int N = 4;
    matrixMultiplyCublas(M, N, K);
    printf("Matrix multiplication completed successfully");
    return 0;
}