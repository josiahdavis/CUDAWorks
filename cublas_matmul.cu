#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void printMatrix(const float* matrix, int N, const char* name){
    std::cout <<"Matrix " << name << " showing top-left 5x5): " << std::endl;
    for (int i = 0; i < 5 && i < N; i++){
        for (int j = 0; j < 5 && j < N; j++){
            std::cout << matrix[i * N + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

void matrixMultiplyCublas(const int N){
    // Allocate host memory
    float *h_A = new float[N * N];
    float *h_B = new float[N * N];
    float *h_C = new float[N * N];

    // Initialize matrices
    for (int i = 0; i < N * N; i++){
        h_A[i] = (float)rand()/(float)RAND_MAX;
        h_B[i] = (float)rand()/(float)RAND_MAX;
        // h_A[i] = static_cast<float>(rand() / RAND_MAX) * 10.0f;
        // h_B[i] = static_cast<float>(rand() / RAND_MAX) * 10.0f;
    }
    printf("%6.2f, %6.2f, %6.2f ", h_A[0], h_A[1], h_A[2]);
    printf("%6.2f, %6.2f, %6.2f ", h_B[0], h_B[1], h_B[2]);
    std::cout << "h_A[0]: " << h_A[0] << " h_B[0]: " << h_B[0] << std::endl;

    // Print input matrices
    printMatrix(h_A, N, "A");
    printMatrix(h_B, N, "B");

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N*N*sizeof(float));
    hipMalloc((void**)&d_B, N*N*sizeof(float));
    hipMalloc((void**)&d_C, N*N*sizeof(float));

    // Copy data to device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // cuBLAS handle is required to use cuBLAS. It initializes the library 
    // and acts as a context for making cuBLAS function calls.
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f, beta = 0.0f;
    hipblasSgemm(handle,HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);

    // Copy results back to host
    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
    
    // Print output matrix
    printMatrix(h_C, N, "C");

    // Clean up
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
}

int main() {
    int N = 512;
    matrixMultiplyCublas(N);
    std::cout << "Matrix multiplication completed successfully" << std::endl;
    return 0;
}