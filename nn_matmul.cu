#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

// He i.e., normal initialization
__global__ void init_rand(int w, int h, float* mat)
{
    int column = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;

    if (row < h && column < w){
        hiprandState state;
        hiprand_init(42, row * w + column, 0, &state);
        mat[row*w + column] = hiprand_normal(&state)*sqrtf(2.f/h);
    }
}

// Computes output = W * X + b
__global__ void matmul(int batch_size, int in_w, int out_w, float* input, 
                        float* weights, float* biases, float* output)
{
    int column = blockIdx.x*blockDim.x + threadIdx.x; 
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < batch_size && column < out_w)
    {
        output[row*out_w + column] = biases[column];
        for(int i = 0; i < in_w; i++)
        {
            output[row * out_w + column] += weights[i * out_w + column] * input[row * in_w + i];
        }
    }
}

void print_matrix(const float* matrix, int rows, int cols, const char* name){
    printf("Matrix %s showing top-left 5x5\n", name);
    for (int i = 0; i < 5 && i < rows; i++){
        for (int j = 0; j < 5 && j < cols; j++){
            printf("%5.2f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(){

    // Set up parameters
    int batch_size = 2;
    int in_features = 3;
    int out_features = 4;
    int BLOCK_SIZE = 16;
    
    // Allocate memory for GPU
    float *d_X;
    float *d_weights;
    float *d_biases;
    float *d_out;
    hipMalloc((void **)&d_X, batch_size * in_features * sizeof(float));
    hipMalloc((void **)&d_weights, in_features * out_features * sizeof(float));
    hipMalloc((void **)&d_biases, out_features * sizeof(float));
    hipMalloc((void **)&d_out, batch_size * out_features * sizeof(float));

    // Initialize weights GPU
    dim3 dimGrid = dim3(ceil(out_features/(float)BLOCK_SIZE), ceil(in_features/(float)BLOCK_SIZE), 1);
    dim3 dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
    init_rand<<<dimGrid, dimBlock>>>(out_features, in_features, d_weights);
    
    // Initialize biases GPU
    dimGrid = dim3(ceil(out_features/(float)BLOCK_SIZE), 1, 1);
    dimBlock = dim3(BLOCK_SIZE, 1, 1);
    init_rand<<<dimGrid, dimBlock>>>(out_features, 1, d_biases);

    // Initialize data GPU
    dimGrid = dim3(ceil(in_features/(float)BLOCK_SIZE), ceil(batch_size/(float)BLOCK_SIZE), 1);
    dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
    init_rand<<<dimGrid, dimBlock>>>(in_features, batch_size, d_X);

    // Perform Matrix Multiplication
    dimGrid = dim3(ceil(out_features/(float)BLOCK_SIZE), ceil(batch_size/(float)BLOCK_SIZE), 1);
    dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
    // batch_size, n, out_w, input, weights, biases, output
    matmul<<<dimGrid, dimBlock>>>(batch_size, in_features, out_features, d_X, d_weights, d_biases, d_out);

    // Copy data to CPU
    float *h_X = new float[batch_size * in_features];
    float *h_weights = new float[in_features * out_features];
    float *h_out = new float[batch_size * out_features];
    float *h_biases = new float[out_features];
    hipMemcpy(h_X, d_X, batch_size * in_features * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_weights, d_weights, in_features * out_features * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_biases, d_biases, out_features * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_out, d_out, batch_size * out_features * sizeof(float), hipMemcpyDeviceToHost);

    // Print data for inspection
    print_matrix(h_X, batch_size, in_features, "X");
    print_matrix(h_weights, in_features, out_features, "Weights");
    print_matrix(h_biases, 1, out_features, "Biases");
    print_matrix(h_out, batch_size, out_features, "Output");
    return 0;
}