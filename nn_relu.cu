#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void init_matrix(float* mat, int rows, int cols){
    int row_i = blockIdx.x * blockDim.x + threadIdx.x;
    int col_j = blockIdx.y * blockDim.y + threadIdx.y;

    if (row_i < rows && col_j < cols){
        int index = row_i * rows + col_j;
        hiprandState state;
        hiprand_init(123, index, 0, &state);
        mat[index] = hiprand_normal(&state)*sqrtf(2.f/rows);
    }
}

__global__ void relu(float* mat_in, float* mat_out, int rows, int cols){
    int row_i = blockIdx.x * blockDim.x + threadIdx.x;
    int col_j = blockIdx.y * blockDim.y + threadIdx.y;

    if (row_i < rows && col_j < cols){
        int index = row_i * rows + col_j;
        float activation = mat_in[index];
        mat_out[index] = activation > 0.f ? activation : 0.f;
    }
}

void print_matrix(const float* matrix, int rows, int cols, const char* name){
    printf("Matrix %s showing top-left 5x10\n", name);
    for (int i = 0; i < 5 && i < rows; i++){
        for (int j = 0; j < 10 && j < cols; j++){
            printf("%5.2f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
    printf("\n");
}


int main(){
    int BLOCK_SIZE = 16;
    
    // Set up parameters
    int batch_size = 4;
    int n_features = 6;
    
    // Allocate memory for GPU
    float *d_in;
    float *d_out;
    hipMalloc((void **)&d_in, batch_size * n_features * sizeof(float));
    hipMalloc((void **)&d_out, batch_size * n_features * sizeof(float));

    // Initialize weights GPU
    dim3 dimGrid = dim3(ceil(batch_size/(float)BLOCK_SIZE), ceil(n_features/(float)BLOCK_SIZE), 1);
    dim3 dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
    init_matrix<<<dimGrid, dimBlock>>>(d_in, batch_size, n_features);
    relu<<<dimGrid, dimBlock>>>(d_in, d_out, batch_size, n_features);
    
    // Copy to CPU
    float *h_in = new float[batch_size * n_features];
    float *h_out = new float[batch_size * n_features];
    hipMemcpy(h_in, d_in, batch_size * n_features * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_out, d_out, batch_size * n_features * sizeof(float), hipMemcpyDeviceToHost);

    // Inspect
    print_matrix(h_in, batch_size, n_features, "Input");
    print_matrix(h_out, batch_size, n_features, "Output");
    return 0;
}
